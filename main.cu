
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
using namespace std;
#define BLOCK_SIZE 32
#define BLOCK_STRIDE 32

const int MAX = 4096;
float* gauss;
float* gauss_h;
//GaussDePyramid[layer][S+2][len][len]

int n;
int gauss_size(int n){
    int layer=0;
    int len=n;
    int length = 0;
    while (n){
        layer++;
        n/=2;
        length += pow(n, 2) * 5;
    }
    return length;
}




__global__ void gauss_de_pyramid_initialize(int length,float* gauss){
    int threadId = blockIdx.x * blockDim.x * blockDim.y * blockDim.z
                   + threadIdx.z * blockDim.y * blockDim.x
                   + threadIdx.y * blockDim.x + threadIdx.x;
    if (threadId < length)
        gauss[threadId] =  1.0;
}
__global__ void gauss_de_pyramid_generate(int length, float* gauss){
    const float sigma = 2.0;
    const float PI = 3.1415926;
    const int S = 2;
    int threadId = blockIdx.x * blockDim.x * blockDim.y * blockDim.z
                   + threadIdx.z * blockDim.y * blockDim.x
                   + threadIdx.y * blockDim.x + threadIdx.x;
    //GaussDePyramid[layer][S+2][len][len]
    int layer = 0;
    int len = length;
    while (len){
        layer++;
        len /= 2;
    }
    int my_id = threadId;
    int my_layer = 0;
    len = length;
    if (threadId >= length){
        return;
    }
    while (my_id > (3 + S) * len * len){
        my_id -= (3 + S) * len * len;
        len /= 2;
        my_layer ++;
    };
    //确定本线程对应的层数
    int my_len = len;

    int my_S = my_id / (len * len);
    //确定本线程对应的S

    int my_r = my_id % (len * len) / len;
    //确定本线程对应的行

    int my_c = my_id % (len * len) % len;
    //确定本线程对应列

    float sig=sigma/(my_S+1);
    gauss[threadId] *= exp(-(my_r-len)*(my_r-len)/(2*sig*sig))/(sig*sqrt(2*PI));
    gauss[threadId] *= exp(-(my_c-len)*(my_c-len)/(2*sig*sig))/(sig*sqrt(2*PI));
    //进行滤波操作

    __syncthreads();
    //同步，以防后续操作出错
    //后续将进行层间差分操作，为了保证一致性进行划分操作

    if(threadId >= layer)
        return;

    len = length;
    while (my_layer){
        my_id += (3 + S) * len * len;
        len /= 2;
        my_layer --;
    }
    //获取当前起始位置

    for(int s = 0; s < (S + 2); s ++)
        for(int i = 0; i < len; i ++)
            for(int j = 0; j < len; j ++){
                gauss[my_id + s*len*len + i*len + j] -= gauss[my_id + (s+1)*len*len + i*len + j];
            }
    __syncthreads();
}
//验证程序正确性
//void output(float* gauss_h_inner,int len){
//    const int S = 2;
//    int layer = 0;
//    int l = len;
//    while (l){
//        l /= 2;
//        layer ++;
//    }
//    int start_id =0;
//    for (int i=0; i<layer; i++){
//        for (int k=0; k < len; k++){
//            for (l=0; l< len; l++){
//                cout<<gauss_h_inner[start_id + k*len + l]<<" ";
//            }
//            cout<<endl;
//        }
//        for (int k=0; k<len; k++){
//            cout<<"==";
//        }
//        cout<<endl;
//        start_id += i*(S+3)*len*len;
//        len /= 2;
//    }
//}
int main() {
    n = 8;
    int deviceId;
    hipGetDevice(&deviceId);
    cout << "deviceId: "<<deviceId<<endl;
    size_t threads_per_block = BLOCK_SIZE;
    size_t number_of_blocks = (n + threads_per_block -1)/threads_per_block;
    while(n < MAX){
        hipHostMalloc(&gauss_h, gauss_size(n), hipHostMallocDefault);
        hipMalloc(&gauss, gauss_size(n));
        std::chrono::duration<double, std::milli> elapsed{};
        auto start= std::chrono::high_resolution_clock::now();
        auto end = std::chrono::high_resolution_clock::now();
        elapsed +=end-start;
        int times =0;
        while (elapsed.count() < 5000){
            gauss_de_pyramid_initialize<<<number_of_blocks*BLOCK_STRIDE,threads_per_block>>>(gauss_size(n),gauss);
            start= std::chrono::high_resolution_clock::now();
            gauss_de_pyramid_generate<<<number_of_blocks*BLOCK_STRIDE,threads_per_block>>>(gauss_size(n),gauss);
            hipDeviceSynchronize();
            end = std::chrono::high_resolution_clock::now();
            elapsed += end-start;
            hipMemcpy(gauss_h,gauss, gauss_size(n),hipMemcpyDeviceToHost);
            hipFree(&gauss);
            hipHostFree(&gauss_h);
            times += 1;
        }
        cout<<n<<","<<float (elapsed.count())/float (times)<<endl;
        n *= 2;
    }
    return 0;
}
